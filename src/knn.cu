#include "hip/hip_runtime.h"
/**
 * 3460:677 Final Project: Paralelizing OCR using PCA
 * Christopher Stoll, 2014
 */

#ifndef KNN_C
#define KNN_C

#include "ocrKit.c"
#include <sys/time.h>
#include <cutil.h>

#define TRAINING_SET_SIZE = 78
#define DEBUG_PRINT_TIME 1

__global__ void nearestNeighborGPUa(int g_klimit, int g_dimensionality, double *g_charWeights, double *g_qWeights, double *g_scores)
{
	extern __shared__ double s_qWeights[];
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	// load shared memory
	if (idx < g_klimit) {
		s_qWeights[idx] = g_qWeights[idx];
	}
	__syncthreads();

	// set some rgisters
	int charWeightIndex = 0;
	int charWeightIndexPart = (idx * (g_dimensionality-1));
	double numerator = 0;
	double denominatorA = 0;
	double denominatorB = 0;
	double totalScore = 0;

	// calulate the cosine similarity
	for (int j = 0; j < g_klimit; ++j) {
		charWeightIndex = charWeightIndexPart + j;

		numerator += s_qWeights[j] * g_charWeights[charWeightIndex];
		denominatorA += s_qWeights[j] * s_qWeights[j];
		denominatorB += g_charWeights[charWeightIndex] * g_charWeights[charWeightIndex];
	}

	if (denominatorA && denominatorB) {
		totalScore = numerator / (sqrt(denominatorA) * sqrt(denominatorB));
	}

	__syncthreads(); 

	// save cosine similarity score
	g_scores[idx] = totalScore;
}

static char launchNearestNeighborA(struct OCRkit *ocrKit, double *questionWeights)
{
	// gather basic information
	int characterCount = ocrKit->characterCount;
	int klimit = (ocrKit->klimit / 4);
	int dimensionality = ocrKit->dimensionality;
	// We are using a static declaration to get more shared memory
	// A pitfall is if the eigen data is updated to include more characters
	// This program must be changed and recompiled
	// if (characterCount != TRAINING_SET_SIZE) {
	// 	printf("WARNING: Character set size %d differs from expected %d\n", characterCount, TRAINING_SET_SIZE);
	// }

	// allocate memory for character weights
	int cWeightMemSize = klimit * dimensionality * sizeof(double);
	double *d_charWeights = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_charWeights, cWeightMemSize));
	CUDA_SAFE_CALL(hipMemcpy(d_charWeights, ocrKit->characterWeights, cWeightMemSize, hipMemcpyHostToDevice));

	// allocate memory for candidate weights
	int qWeightMemSize = dimensionality * sizeof(double);
	double *d_qWeights = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_qWeights, qWeightMemSize));
	CUDA_SAFE_CALL(hipMemcpy(d_qWeights, questionWeights, qWeightMemSize, hipMemcpyHostToDevice));

	// allocate memory for scores
	int scoreMemSize = characterCount * sizeof(double);
	double *h_scores = (double*)malloc(scoreMemSize);
	memset(h_scores, 0, characterCount * sizeof(double));
	double *d_scores = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_scores, scoreMemSize));
	CUDA_SAFE_CALL(hipMemcpy(d_scores, h_scores, scoreMemSize, hipMemcpyHostToDevice));

	// set up parallel dimensions
	int threadsPerBlock = characterCount;
	int blocksPerGrid = 1;
	dim3 dimGrid(blocksPerGrid);
	dim3 dimBlock(threadsPerBlock);
	int sharedMemSize = klimit * sizeof(double);

	// run the kernel
	nearestNeighborGPUa<<< dimGrid,dimBlock,sharedMemSize >>>(klimit, dimensionality, d_charWeights, d_qWeights, d_scores);
	hipDeviceSynchronize();

	// get the scores
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, scoreMemSize, hipMemcpyDeviceToHost));

	char answer = '?';
	int maxScore = -999;
	for (int i = 0; i < characterCount; ++i) {
		if (h_scores[i] > maxScore) {
			maxScore = h_scores[i];
			answer = ocrKit->characters[i];
		}
	}
	return answer;
}

static char nearestNeighbor(struct OCRkit *ocrKit, double *questionWeights)
{
	struct timeval stop, start;
	char answer = '?';

	gettimeofday(&start, NULL);
	//answer = nearestNeighborCPU(ocrKit, questionWeights);
	answer = launchNearestNeighborA(ocrKit, questionWeights);
	gettimeofday(&stop, NULL);

	if (DEBUG_PRINT_TIME) {
		printf("Time: %u us (%c)\n", (unsigned int)(stop.tv_usec - start.tv_usec), answer);
	}

	return answer;
}

#endif
