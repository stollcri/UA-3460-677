#include "hip/hip_runtime.h"
/**
 * 3460:677 Final Project: Paralelizing OCR using PCA
 * Christopher Stoll, 2014
 */

#ifndef KNN_C
#define KNN_C

#include "ocrKit.c"
#include <sys/time.h>
#include <cutil.h>

#define TRAINING_SET_SIZE = 78
#define DEBUG_PRINT_TIME 1

static void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if(hipSuccess != err)  {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		exit(EXIT_FAILURE);
	}
}

__global__ void nearestNeighborGPUa(int g_klimit, int g_dimensionality, double *g_charWeights, double *g_qWeights, double *g_scores)
{
	// extern __shared__ double s_qWeights[];
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	// // load shared memory
	// if (idx < g_dimensionality) {
	// 	s_qWeights[idx] = g_qWeights[idx];
	// }
	__syncthreads();

	// set some rgisters
	int charWeightIndex = 0;
	int charWeightIndexPart = (idx * (g_dimensionality-1));
	double numerator = 0;
	double denominatorA = 0;
	double denominatorB = 0;
	double totalScore = 0;

	// calulate the cosine similarity
	for (int j = 0; j < g_klimit; ++j) {
		charWeightIndex = charWeightIndexPart + j;
		
		numerator += g_qWeights[j] * g_charWeights[charWeightIndex];
		denominatorA += g_qWeights[j] * g_qWeights[j];
		denominatorB += g_charWeights[charWeightIndex] * g_charWeights[charWeightIndex];
	}

	if (denominatorA && denominatorB) {
		totalScore = numerator / (sqrt(denominatorA) * sqrt(denominatorB));
	}

	// save cosine similarity score
	g_scores[idx] = totalScore;
}

static char launchNearestNeighborA(struct OCRkit *ocrKit, double *questionWeights)
{
	// gather basic information
	int characterCount = ocrKit->characterCount;
	int klimit = (ocrKit->klimit / 4);
	int dimensionality = ocrKit->dimensionality;
	int questionWeightsCount = ocrKit->imageDoc->totalWeightCount;
	// We are using a static declaration to get more shared memory
	// A pitfall is if the eigen data is updated to include more characters
	// This program must be changed and recompiled
	// if (characterCount != TRAINING_SET_SIZE) {
	// 	printf("WARNING: Character set size %d differs from expected %d\n", characterCount, TRAINING_SET_SIZE);
	// }

	// allocate memory for character weights
	int cWeightMemSize = klimit * dimensionality * sizeof(double);
	double *d_charWeights = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_charWeights, cWeightMemSize));
	CUDA_SAFE_CALL(hipMemcpy(d_charWeights, ocrKit->characterWeights, cWeightMemSize, hipMemcpyHostToDevice));

	// allocate memory for candidate weights
	int qWeightMemSize = questionWeightsCount * sizeof(double);
	double *d_qWeights = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_qWeights, qWeightMemSize));
	CUDA_SAFE_CALL(hipMemcpy(d_qWeights, questionWeights, qWeightMemSize, hipMemcpyHostToDevice));

	// allocate memory for scores
	int scoreMemSize = characterCount * sizeof(double);
	double *h_scores = (double*)malloc(scoreMemSize);
	double *d_scores = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_scores, scoreMemSize));

	// set up parallel dimensions
	int threadsPerBlock = characterCount;
	int blocksPerGrid = questionWeightsCount / klimit;
	dim3 dimGrid(blocksPerGrid);
	dim3 dimBlock(threadsPerBlock);
	// int sharedMemSize = dimensionality * sizeof(double);

	// run the kernel
	nearestNeighborGPUa<<< dimGrid,dimBlock >>>(klimit, dimensionality, d_charWeights, d_qWeights, d_scores);
	hipDeviceSynchronize();
	checkCUDAError("kernel");

	// get the scores
	CUDA_SAFE_CALL(hipMemcpy(h_scores, d_scores, scoreMemSize, hipMemcpyDeviceToHost));
	checkCUDAError("memcpy");

	char answer = '?';
	int maxScore = -999;
	for (int i = 0; i < characterCount; ++i) {
		if (h_scores[i] > maxScore) {
			maxScore = h_scores[i];
			answer = ocrKit->characters[i];
		}
	}

	hipFree(d_charWeights);
	hipFree(d_qWeights);
	hipFree(d_scores);
	free(h_scores);

	return answer;
}

void nearestNeighbor(struct OCRkit *ocrKit, double *candidateWeights)
{
	struct timeval stop, start;

	gettimeofday(&start, NULL);
	launchNearestNeighborA(ocrKit, candidateWeights);
	gettimeofday(&stop, NULL);

	if (DEBUG_PRINT_TIME) {
		printf("Time: %u us (%c)\n", (unsigned int)(stop.tv_usec - start.tv_usec), '?');
	}
}

#endif
