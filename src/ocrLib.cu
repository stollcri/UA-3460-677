#include "hip/hip_runtime.h"
/**
 * 3460:677 Final Project: Paralelizing OCR using PCA
 * Christopher Stoll, 2014
 */

#ifndef OCRLIB_C
#define OCRLIB_C

#include <stdlib.h>
#include <math.h>
#include "ocrKit.c"
#include "imageDocument.c"
#include "resizeImage.c"

#define STANDARD_IMAGE_SIDE 16
#define DEBUG_SAVE_STANDARDIZED_CHARACTERS 0
#define DEBUG_PRINT_STANDARDIZED_CHARACTERS 0

static int standardizeImageMatrix(int *imageVector, int imageWidth, struct imageDocumentChar *imageDocChar, int **charImage)
{
	int width = imageDocChar->x2 - imageDocChar->x1;
	int height = imageDocChar->y2 - imageDocChar->y1;

	// ignore things, spaces, which do not need sizing
	if ((width == 0) || height == 0) {
		return 0;
	}

	int padding = 0;
	int paddingQ = 0;
	int paddingR = 0;
	int newWidth = width;
	int newHeight = height;

	int padLeft = 0;
	int padRight = newWidth;
	int padTop = 0;
	int padBottom = newHeight;

	// only if the area is rectangular
	// determine how to fix the shorter dimension
	if (height != width) {
		if (height > width) {
			padding = height - width;
			paddingQ = (int)(padding / 2);
			paddingR = padding - (paddingQ * 2);
			newWidth = paddingQ + width + paddingQ + paddingR;

			padLeft = paddingQ;
			padRight = padLeft + width - 1;
		} else {
			padding = width - height;
			paddingQ = (int)(padding / 2);
			paddingR = padding - (paddingQ * 2);
			newHeight = paddingQ + paddingR + height + paddingQ;

			padTop = paddingQ + paddingR;
			padBottom = padTop + height - 1;
		}
	}
	// create a working image
	int *tempImage = (int*)malloc((unsigned long)newWidth * (unsigned long)newHeight * sizeof(int));
	memset(tempImage, 0, ((unsigned long)newWidth * (unsigned long)newHeight * sizeof(int)));

	int k = 0;
	int l = 0;
	int imagePixel = 0;
	int currentPixel = 0;
	for (int i = 0; i < newHeight; ++i) {
		if ((i >= padTop) && i <= padBottom) {
			k = 0;
			for (int j = 0; j < newWidth; ++j) {
				if ((j >= padLeft) && (j <= padRight)) {
					imagePixel = ((imageDocChar->y1 + l) * imageWidth) + (imageDocChar->x1 + k);
					currentPixel = (i * newWidth) + j;
					++k;
					
					tempImage[currentPixel] = imageVector[imagePixel];
				}
			}
			++l;
		}
	}

	int targetImageWidth = STANDARD_IMAGE_SIDE;
	int *resizedImage = (int*)malloc((unsigned long)targetImageWidth * (unsigned long)targetImageWidth * sizeof(int));
	memset(resizedImage, 0, ((unsigned long)targetImageWidth * (unsigned long)targetImageWidth * sizeof(int)));
	sizeSquareImage(tempImage, resizedImage, newWidth, targetImageWidth);
	
	if (DEBUG_SAVE_STANDARDIZED_CHARACTERS) {
		char fName[100] = "./tst/tst-1-";
		char buffer[32];
		snprintf(buffer, sizeof(buffer), "%d-%d.png", imageDocChar->y1, imageDocChar->x1);
		strcat(fName, buffer);
		write_png_file(resizedImage, targetImageWidth, targetImageWidth, fName);
		// write_png_file(tempImage, newWidth, newHeight, fName);
	}

	*charImage = resizedImage;
	if (DEBUG_PRINT_STANDARDIZED_CHARACTERS) {
		for (int i = 0; i < (STANDARD_IMAGE_SIDE*STANDARD_IMAGE_SIDE); ++i) {
			printf("%3d ", resizedImage[i]);
			if (!(i % STANDARD_IMAGE_SIDE)) {
				printf("\n");
			}
			if (!(i % (STANDARD_IMAGE_SIDE*STANDARD_IMAGE_SIDE))) {
				printf("\n");
			}
		}
	}
	return 1;
}

static double *projectCandidate(int *charImageVector, struct OCRkit *ocrKit)
{
	int klimit = (int)(ocrKit->klimit / 4);
	double *tempWeights = (double*)malloc((unsigned long)klimit * sizeof(double));
	memset(tempWeights, 0, ((unsigned long)klimit * sizeof(double)));

	double *eigenImageSpace = ocrKit->eigenImageSpace;
	int dimensionality = ocrKit->dimensionality;

	int currentEigen = 0;
	double weight = 0;
	for (int i = 0; i < klimit; ++i) {
		weight = 0;
		for (int j = 0; j < (STANDARD_IMAGE_SIDE * STANDARD_IMAGE_SIDE); ++j) {
			currentEigen = (i * dimensionality) + j;
			weight += (charImageVector[j] * eigenImageSpace[currentEigen]);
		}
		tempWeights[i] = weight;
	}
	return tempWeights;
}

static void ocrCharacter(struct OCRkit *ocrKit, struct imageDocumentChar *imageDocChar)
{
	if (imageDocChar) {
		//printf("%c", imageDocChar->value);
		int *charImage;
		int standardizeOk = standardizeImageMatrix(ocrKit->imageVector, ocrKit->imageWidth, imageDocChar, &charImage);
		if (standardizeOk) {
			ocrKit->weights = projectCandidate(charImage, ocrKit);
			free(charImage);

			// char answer = nearestNeighbor(ocrKit, weights);
			// imageDocChar->value = answer;
			// printf("%c", answer);
		// } else {
			// imageDocChar->value = imageDocChar->value;
			// printf("%c", imageDocChar->value);
		}
	}
}

static void ocrCharLoop(struct OCRkit *ocrKit, struct imageDocumentLine *imageDocLine)
{
	if (imageDocLine) {
		if (imageDocLine->characters) {
			struct imageDocumentChar *currentChar = imageDocLine->characters;
			struct imageDocumentChar *nextChar = NULL;

			ocrCharacter(ocrKit, currentChar);

			while (currentChar->nextChar) {
				nextChar = currentChar->nextChar;
				currentChar = nextChar;

				ocrCharacter(ocrKit, currentChar);
			}

			freeImageDocumentChar(nextChar);
		}
	}
}

static void ocrLineLoop(struct OCRkit *ocrKit)
{
	struct imageDocument *imageDoc = ocrKit->imageDoc;
	if (imageDoc) {
		if (imageDoc->lines) {
			struct imageDocumentLine *currentLine = imageDoc->lines;
			struct imageDocumentLine *nextLine = NULL;

			ocrCharLoop(ocrKit, currentLine);

			while (currentLine->nextLine) {
				nextLine = currentLine->nextLine;
				currentLine = nextLine;

				ocrCharLoop(ocrKit, currentLine);
			}

			freeImageDocumentLine(nextLine);
		}
	}
}

static void startOcr(struct OCRkit *ocrKit)
{
	ocrLineLoop(ocrKit);
	nearestNeighbor(ocrKit, vectorizeWeights(ocrKit->imageDoc));
}

#endif
